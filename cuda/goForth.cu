#include <hip/hip_runtime.h>

#include "../support/squaret.h"
#include "../support/io.h"
#include "goForth.h"
#include "step.cuh"

void goForthAndMultiply(square_t *gridA, square_t *gridB, const long rows,
                        const long cols, const long numSteps) {
  square_t *gridDevA, *gridDevB;
  hipMalloc(&gridDevA, rows*cols);
  hipMalloc(&gridDevB, rows*cols);
  hipMemcpy(gridDevA, gridA, rows*cols, hipMemcpyHostToDevice);
  hipMemcpy(gridDevB, gridB, rows*cols, hipMemcpyHostToDevice);

  dim3 dimBlock(rows/32 + 1, cols/32 + 1);
  dim3 dimGrid(32, 32);
  printGrid(gridA, rows, cols);
  for(long i = 0; i < numSteps; i++) {
    step<<<dimBlock, dimGrid>>>(gridDevA, gridDevB, rows, cols);
    square_t *temp = gridDevA;
    gridDevA = gridDevB;
    gridDevB = temp;
  }

  hipMemcpy(gridA, gridDevA, rows*cols, hipMemcpyDeviceToHost);
  printGrid(gridA, rows, cols);

  hipFree(gridDevA);
  hipFree(gridDevB);
}