#include <hip/hip_runtime.h>

#include "../support/squaret.h"
#include "goForth.h"
#include "step.cuh"

void goForthAndMultiply(square_t *gridA, square_t *gridB, const long rows,
                        const long cols, const long numSteps) {
  square_t *gridDevA, *gridDevB;
  hipMalloc(&gridDevA, rows*cols);
  hipMalloc(&gridDevB, rows*cols);
  hipMemcpy(&gridDevA, &gridA, rows*cols, hipMemcpyHostToDevice);
  hipMemcpy(&gridDevB, &gridB, rows*cols, hipMemcpyHostToDevice);

  dim3 dims(cols, rows);

  for(long i = 0; i < numSteps; i++) {
    step<<<1,dims>>>(gridDevA, gridDevB, rows, cols);
  }

  hipMemcpy(&gridA, &gridDevA, rows*cols, hipMemcpyDeviceToHost);
  hipMemcpy(&gridB, &gridDevB, rows*cols, hipMemcpyDeviceToHost);
  hipFree(&gridDevA);
  hipFree(&gridDevB);
}